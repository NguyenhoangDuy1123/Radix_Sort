#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

__global__ void histogram(unsigned int* in, unsigned int* hist, int n,unsigned int nBins, unsigned int mask, unsigned int current_bits)
{
	extern __shared__ unsigned int s_local_hist[];
	
	for(int j = threadIdx.x; j < nBins; j += blockDim.x)
		s_local_hist[j] = 0;
	
	__syncthreads();
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		unsigned int bin = (in[i] >> current_bits) & mask;
		atomicAdd(&s_local_hist[bin], 1);
	}
	__syncthreads();
	
	for (int bin = threadIdx.x; bin < nBins; bin += blockDim.x)
	{
		hist[bin * gridDim.x + blockIdx.x] = s_local_hist[bin];
	}
}

__global__ void scanBlks(unsigned int *in, unsigned int *out, unsigned int n, unsigned int *blkSums)
{

	extern __shared__ int blkData[];
	int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	int i2 = i1 + blockDim.x;
	if (i1 < n)
		blkData[threadIdx.x] = in[i1];
	if (i2 < n)
		blkData[threadIdx.x + blockDim.x] = in[i2];
	__syncthreads();


	for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
	{
		int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1; 
		if (blkDataIdx < 2 * blockDim.x)
			blkData[blkDataIdx] += blkData[blkDataIdx - stride];
		__syncthreads();
	}

	for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride; 
		if (blkDataIdx < 2 * blockDim.x)
			blkData[blkDataIdx] += blkData[blkDataIdx - stride];
		__syncthreads();
	}


	if (i1 < n)
		out[i1] = blkData[threadIdx.x];
	if (i2 < n)
		out[i2] = blkData[threadIdx.x + blockDim.x];

	if (blkSums != NULL && threadIdx.x == 0)
		blkSums[blockIdx.x] = blkData[2 * blockDim.x - 1];

}

__global__ void addPrevSum(unsigned int* blkSumsScan, unsigned int* blkScans, unsigned int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x;
	if (i < n)
	{
		blkScans[i] += blkSumsScan[blockIdx.x];
	}
}

void scanAll(unsigned int *d_in, unsigned int *d_out, unsigned int n, unsigned int blkSize, unsigned int blkDataSize)
{
	if (n <= blkDataSize)
	{
		scanBlks<<<1, blkSize, blkDataSize * sizeof(int)>>>(d_in, d_out, n, NULL);
		hipDeviceSynchronize();
	}
	else
	{
		unsigned int *d_blkSums;
		unsigned int numBlks = (n - 1) / blkDataSize + 1;
		hipMalloc(&d_blkSums, numBlks * sizeof(unsigned int));
		scanBlks<<<numBlks, blkSize, blkDataSize * sizeof(unsigned int)>>>(d_in, d_out, n, d_blkSums);
		hipDeviceSynchronize();
		
		scanAll(d_blkSums, d_blkSums, numBlks, blkSize, blkDataSize);
		
		addPrevSum<<<numBlks - 1, blkDataSize>>>(d_blkSums, d_out, n);
		hipDeviceSynchronize();
	}	
}

__global__ void exclusive_scan(unsigned int *in,unsigned int *out, int n)
{   
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

 	if (i < n)
 	{
		out[i] -= in[i];
	}
}

__global__ void scatter(unsigned int *in,unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int *d_histScan, unsigned int mask, unsigned int current_bits, unsigned int nBins)
{
	extern __shared__ unsigned int min_Idx[];
	
	for(int j = threadIdx.x; j < nBins; j += blockDim.x)
		min_Idx[j] = n;
	__syncthreads();
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		unsigned int bin = (in[i] >> current_bits) & mask;
		atomicMin(&min_Idx[bin], i);
	}
	__syncthreads();
	
	if(i < n)
	{
		unsigned int bin = (in[i] >> current_bits) & mask;
		out[d_histScan[blockIdx.x + bin*gridDim.x] + i - min_Idx[bin]] = in[i];
		out_pos[d_histScan[blockIdx.x + bin*gridDim.x] + i - min_Idx[bin]] = in_pos[i];
	}
}

__global__ void swap(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < n)
	{
		in[i] = in[i] ^ out[i];
		out[i] = in[i] ^ out[i];
		in[i] = in[i] ^ out[i];
		
		in_pos[i] = in_pos[i] ^ out_pos[i];
		out_pos[i] = in_pos[i] ^ out_pos[i];
		in_pos[i] = in_pos[i] ^ out_pos[i];
	}
}

__global__ void pre_sort(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int nBins, unsigned int mask, unsigned int current_bits, unsigned int *d_hist)
{
	extern __shared__ unsigned int pre_sort_blk_data[];
	unsigned int* blk_value = pre_sort_blk_data;
	unsigned int* blk_pos = pre_sort_blk_data + blockDim.x;
	unsigned int* blk_hist = pre_sort_blk_data + 2*blockDim.x;
	unsigned int* blk_Scan = pre_sort_blk_data + nBins + 2*blockDim.x;
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		blk_value[threadIdx.x] = in[i];
		blk_pos[threadIdx.x] = in_pos[i];
	}
	__syncthreads();
	
	//Hist
	for(int j = threadIdx.x; j < nBins; j += blockDim.x)
	{
		blk_hist[j] = 0;
		blk_Scan[j] = 0;
	}
	__syncthreads();
	
	unsigned int bin = (blk_value[threadIdx.x] >> current_bits) & mask;
	atomicAdd(&blk_hist[bin], 1);
	atomicAdd(&blk_Scan[bin], 1);
	__syncthreads();
	
	//Scan
	for (int stride = 1; stride < nBins; stride *= 2)
	{	
		for (int k = threadIdx.x; k < nBins; k += blockDim.x)
		{
			int inVal;
			if (k >= stride)
				inVal = blk_Scan[k - stride];
			__syncthreads();
			if (k >= stride)
				blk_Scan[k] += inVal;
			__syncthreads();
		}
	}
	__syncthreads();
	
	for (int i = threadIdx.x; i < nBins; i += blockDim.x)
		blk_Scan[i] -= blk_hist[i];
	__syncthreads();
	
	//Scatter
	if (threadIdx.x == 0)
	{
		for (int i = 0; i < blockDim.x; i++)
		{
			unsigned int bin = (blk_value[i] >> current_bits) & mask;
			out[blk_Scan[bin] + blockIdx.x*blockDim.x] = blk_value[i];
			out_pos[blk_Scan[bin] + blockIdx.x*blockDim.x] = blk_pos[i];
			blk_Scan[bin]++;
		}
	}
}

const dim3 hist_blockSize(257);
const dim3 scan_blockSize(512);
const dim3 swap_blockSize(1024);

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
    unsigned int nBits = 8;
	
	unsigned int nBins = 1 << nBits;
		
	dim3 hist_gridSize((numElems - 1)/(hist_blockSize.x) + 1);
	dim3 scan_gridSize((hist_gridSize.x * nBins - 1)/(scan_blockSize.x) + 1);
	dim3 swap_gridSize((numElems - 1)/(swap_blockSize.x) + 1);
	
    unsigned int *d_hist;
	hipMalloc(&d_hist, hist_gridSize.x * nBins * sizeof(unsigned int));
	
	unsigned int *d_histScan;
	hipMalloc(&d_histScan, hist_gridSize.x * nBins * sizeof(unsigned int));
	
	unsigned int mask = (1 << nBits) - 1;
	
    for (unsigned int i = 0; i < sizeof(unsigned int)*8; i += nBits)
    {
		//Pre_sort
		pre_sort<<<hist_gridSize, hist_blockSize, (2*hist_blockSize.x + 2*nBins)*sizeof(unsigned int)>>>(d_inputVals, d_inputPos, d_inputVals, d_inputPos, numElems, nBins, mask, i, d_hist);

		//Histogram
		histogram<<<hist_gridSize, hist_blockSize, nBins*sizeof(unsigned int)>>>(d_inputVals, d_hist, numElems, nBins, mask, i);
	
		//Exclusive Scan
		scanAll(d_hist, d_histScan, hist_gridSize.x * nBins, scan_blockSize.x, 2*scan_blockSize.x);
		exclusive_scan<<<scan_gridSize, scan_blockSize>>>(d_hist, d_histScan, hist_gridSize.x * nBins);

		//Scatter
		scatter<<<hist_gridSize, hist_blockSize, nBins*sizeof(unsigned int)>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems, d_histScan, mask, i, nBins);
		
		//Swap
		swap<<<swap_gridSize, swap_blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);
    }
	swap<<<swap_gridSize, swap_blockSize>>>(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);
}