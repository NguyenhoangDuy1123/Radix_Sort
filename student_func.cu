#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"
#include <iostream>
#include <stdio.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.
   Note: ascending order == smallest to largest
   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.
   Implementing Parallel Radix Sort with CUDA
   ==========================================
   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.
   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there
   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.
 */

__global__ void histogram(unsigned int* in, unsigned int* hist, int n,unsigned int nBins, unsigned int mask, unsigned int current_bits)
{
	extern __shared__ unsigned int s_local_hist[];
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	for(int j = threadIdx.x; j < nBins; j += blockDim.x)
		s_local_hist[j] = 0;
	__syncthreads();
	
	if (i < n)
	{
		unsigned int bin = (in[i] >> current_bits) & mask;
		atomicAdd(&s_local_hist[bin], 1);
	}
	__syncthreads();
	
	for (unsigned int bin = threadIdx.x; bin < nBins; bin += blockDim.x)
		atomicAdd(&hist[bin], s_local_hist[bin]);
}

__global__ void scan(unsigned int *in,unsigned int *out, int n)
{   
	extern __shared__ int blkData[];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		blkData[threadIdx.x] = in[i];
	__syncthreads();

	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		int left_value;
		if (threadIdx.x >= stride)
			left_value = blkData[threadIdx.x - stride];
		__syncthreads();
		
		if (threadIdx.x >= stride)
			blkData[threadIdx.x] += left_value;
		__syncthreads();
	}

	if (i < n)
		if(i == 0)
			out[i] = 0;
		else
			out[i] = blkData[i - 1];
}

const dim3 hist_blockSize(256);
const dim3 scan_blockSize(256);

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
    unsigned int nBits;
	for (int i = 1; i < 32; i++)
		if (1 << i > scan_blockSize.x)
		{
			nBits = i - 1;
			break;
		}
	
	dim3 hist_gridSize((numElems - 1)/(hist_blockSize.x) + 1);
    unsigned int nBins = 1 << nBits;
    unsigned int *d_hist;
	hipMalloc(&d_hist, nBins * sizeof(unsigned int));
	
    unsigned int* histScan = new unsigned int[nBins];
	unsigned int *d_histScan;
	hipMalloc(&d_histScan, nBins * sizeof(unsigned int));
    
    unsigned int* src = new unsigned int[numElems];
    unsigned int* src_pos = new unsigned int[numElems];
    
    unsigned int* des = new unsigned int[numElems];
    unsigned int* des_pos = new unsigned int[numElems];
    
    hipMemcpy(src, d_inputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(src_pos, d_inputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
	unsigned int *d_src;
	hipMalloc(&d_src, numElems * sizeof(unsigned int));
	
	unsigned int mask = (1 << nBits) - 1;
    for (unsigned int i = 0; i < sizeof(unsigned int)*8; i += nBits)
    {
		hipMemcpy(d_src, src, numElems * sizeof(unsigned int), hipMemcpyHostToDevice);
		//Histogram
		hipMemset(d_hist, 0, nBins * sizeof(unsigned int));
        
		histogram<<<hist_gridSize, hist_blockSize, nBins*sizeof(unsigned int)>>>(d_src, d_hist, numElems, nBins, mask, i);
		
		hipDeviceSynchronize();
		 
		//Exclusive Scan
		scan<<<1, scan_blockSize, scan_blockSize.x*sizeof(unsigned int)>>>(d_hist, d_histScan, nBins);
		hipMemcpy(histScan, d_histScan, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);
		
		hipDeviceSynchronize();
            
		// Scatter
        for (unsigned int j = 0; j < numElems; j++)
        {
            unsigned int bin = (src[j] >> i) & mask;
            des[histScan[bin]] = src[j];
			des_pos[histScan[bin]] = src_pos[j];
            histScan[bin]++;
        }
        
		//Swap
        for (unsigned int j = 0; j < numElems; j++)
        {
            unsigned int temp = src[j];
            src[j] = des[j];
            des[j] = temp;
			
			temp = src_pos[j];
            src_pos[j] = des_pos[j];
            des_pos[j] = temp;
        }
    }
	
	hipMemcpy(d_outputVals, src, numElems * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_outputPos, src_pos, numElems * sizeof(unsigned int), hipMemcpyHostToDevice);
    
    delete[] histScan;
    delete[] src;
    delete[] src_pos;
    delete[] des;
    delete[] des_pos;
}